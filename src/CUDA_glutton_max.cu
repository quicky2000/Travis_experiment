#include "hip/hip_runtime.h"
/*
      This file is part of edge_matching_puzzle
      Copyright (C) 2021  Julien Thevenon ( julien_thevenon at yahoo.fr )

      This program is free software: you can redistribute it and/or modify
      it under the terms of the GNU General Public License as published by
      the Free Software Foundation, either version 3 of the License, or
      (at your option) any later version.

      This program is distributed in the hope that it will be useful,
      but WITHOUT ANY WARRANTY; without even the implied warranty of
      MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
      GNU General Public License for more details.

      You should have received a copy of the GNU General Public License
      along with this program.  If not, see <http://www.gnu.org/licenses/>
*/

#include "emp_FSM_info.h"
#include "emp_piece_db.h"
#include "CUDA_glutton_max_stack.h"
#include "CUDA_color_constraints.h"
#include "CUDA_memory_managed_array.h"
#include "my_cuda.h"
#include "CUDA_common.h"
#include "emp_situation.h"
#include "situation_string_formatter.h"

#define LOG_EXECUTION

namespace edge_matching_puzzle
{

    /**
     * Store piece representation.
     * First dimension is piece index ( ie piece id -1 )
     * Second dimension is border orientation
     */
    __constant__ uint32_t g_pieces[256][4];

    /**
     * Return position offset for each orientation
     * NORTH : 0 EAST:1 SOUTH:2 WEST:3
     * Position offset depend on puzzle dimensions
     */
    __constant__ int g_position_offset[4];

    /**
     * Number of pieces remaining to set
     */
    __constant__ unsigned int g_nb_pieces;

    __device__
    unsigned int my_strlen(const char * p_string)
    {
        unsigned int l_index = 0;
        while('\0' != p_string[l_index])
        {
            ++l_index;
        }
        return l_index;
    }

    __device__
    unsigned int count_cr(const char * p_string)
    {
        unsigned int l_nb_cr = 0;
        unsigned int l_index = 0;
        while('\0' != p_string[l_index])
        {
            l_nb_cr += ('\n' == p_string[l_index++]);
        }
        return l_nb_cr;
    }

    __device__
    char * prepare_format(unsigned int p_level
                         ,const char * p_header
                         ,const char * p_format
                         )
    {
        unsigned int l_len = my_strlen(p_format);
        unsigned int l_nb_cr = count_cr(p_format);
        bool l_additional_cr = (!l_len) || (p_format[l_len - 1] != '\n');
        unsigned int l_total_cr = l_nb_cr + l_additional_cr;
        unsigned int l_header_len = my_strlen(p_header);
        char * l_format = static_cast<char*>(malloc(l_len + l_additional_cr + l_total_cr * (l_header_len + 2 * p_level)));
        unsigned int l_index = 0;
        unsigned int l_char_index = 0;
        for(unsigned int l_line_index = 0; l_line_index < l_total_cr; ++l_line_index)
        {
            for(unsigned int l_level_index = 0; l_level_index < 2 * p_level; ++l_level_index)
            {
                l_format[l_index++] = ' ';
            }
            for(unsigned int l_header_index = 0; l_header_index < l_header_len; ++l_header_index)
            {
                l_format[l_index++] = l_line_index ? ' ' : p_header[l_header_index];
            }
            while('\0' != p_format[l_char_index] && '\n' != p_format[l_char_index])
            {
                l_format[l_index++] = p_format[l_char_index++];
            }
            ++l_char_index;
            l_format[l_index++] = '\n';
        }
        l_format[l_index++] = '\0';
        return l_format;
    }

    template<typename... Targs>
    __device__
    void print_all(unsigned int p_level
                  ,const char * p_format
                  ,Targs... Fargs
                  )
    {
#ifdef LOG_EXECUTION
        char * l_format = prepare_format(p_level, "Thread%3i : ", p_format);
        printf(l_format, threadIdx.x, Fargs...);
        free(l_format);
#endif // LOG_EXECUTION
    }

    template<typename... Targs>
    __device__
    void print_mask(unsigned int p_level
                   ,uint32_t p_mask
                   ,const char * p_format
                   ,Targs... Fargs
                   )
    {
#ifdef LOG_EXECUTION
        // Do format treatment in all threads to minimise divergence
        char * l_format = prepare_format(p_level, "Thread%3i : ", p_format);
        if((1u << threadIdx.x) & p_mask)
        {
            printf(l_format, threadIdx.x, Fargs...);
        }
        free(l_format);
#endif // LOG_EXECUTION
    }

    template<typename... Targs>
    __device__
    void print_single(unsigned int p_level
                     ,const char * p_format
                     ,Targs... Fargs
                     )
    {
#ifdef LOG_EXECUTION
        char * l_format = prepare_format(p_level, "", p_format);
        if(!threadIdx.x)
        {
            printf(l_format, Fargs...);
        }
        free(l_format);
#endif // LOG_EXECUTION
    }

    __global__
    void test_kernel(CUDA_glutton_max_stack * p_stacks
                    ,unsigned int p_nb_stack
                    ,CUDA_memory_managed_array<uint32_t> & p_array
                    );

    inline
    __device__
    uint32_t reduce_add_sync(uint32_t p_word)
    {
        unsigned l_mask = 0xFFFF;
        unsigned int l_width = 16;
        do
        {
            p_word += __shfl_down_sync(l_mask, p_word, l_width);
            l_width = l_width >> 1;
            l_mask = l_mask >> l_width;
        }
        while(l_width);
        return __shfl_sync(0xFFFFFFFFu, p_word, 0);
    }

    inline
    __device__
    uint32_t reduce_min_sync(uint32_t p_word)
    {
        unsigned l_mask = 0xFFFF;
        unsigned int l_width = 16;
        do
        {
            uint32_t l_received_word = __shfl_down_sync(l_mask, p_word, l_width);
            p_word = l_received_word < p_word ? l_received_word : p_word;
            l_width = l_width >> 1;
            l_mask = l_mask >> l_width;
        }
        while(l_width);
        return __shfl_sync(0xFFFFFFFFu, p_word, 0);
    }

    inline
    __device__
    uint32_t reduce_max_sync(uint32_t p_word)
    {
        unsigned l_mask = 0xFFFF;
        unsigned int l_width = 16;
        do
        {
            uint32_t l_received_word = __shfl_down_sync(l_mask, p_word, l_width);
            p_word = l_received_word > p_word ? l_received_word : p_word;
            l_width = l_width >> 1;
            l_mask = l_mask >> l_width;
        }
        while(l_width);
        return __shfl_sync(0xFFFFFFFFu, p_word, 0);
    }

    inline
    __device__
    void update_stats(uint32_t p_value
                     ,uint32_t & p_min
                     ,uint32_t & p_max
                     ,uint32_t & p_total
                     )
    {
        p_max = p_value > p_max ? p_value : p_max;
        p_min = p_value < p_min ? p_value : p_min;
        p_total += p_value;
    }

    inline
    __device__
    bool analyze_info(uint32_t p_capability
                     ,uint32_t p_constraint_capability
                     ,uint32_t & p_min
                     ,uint32_t & p_max
                     ,uint32_t & p_total
                     ,CUDA_glutton_max_stack::t_piece_infos & p_piece_info
                     )
    {
        uint32_t l_result_capability = p_capability & p_constraint_capability;

        // Check result of mask except for selected piece and current position
        if(__any_sync(0xFFFFFFFFu, l_result_capability))
        {
            uint32_t l_info_bits = reduce_add_sync(__popc(l_result_capability));
            update_stats(l_info_bits, p_min, p_max, p_total);
            for(unsigned short & l_piece_index : p_piece_info)
            {
                l_piece_index += static_cast<CUDA_glutton_max_stack::t_piece_info>(__popc(static_cast<int>(l_result_capability & 0xFu)));
                l_result_capability = l_result_capability >> 4;
            }
            return false;
        }
        return true;
    }

    __device__
    void print_position_info(unsigned int p_indent_level
                            ,const CUDA_glutton_max_stack & p_stack
                            ,const CUDA_piece_position_info2 & (CUDA_glutton_max_stack::*p_accessor)(unsigned int) const
                            )
    {
        for(unsigned int l_display_index = 0; l_display_index < (p_stack.get_size() - p_stack.get_level()); ++l_display_index)
        {
            print_all(6,"Info = 0x%" PRIx32, (p_stack.*p_accessor)(l_display_index).get_word(threadIdx.x));
        }
    }
    __device__
    void print_position_info(unsigned int p_indent_level
                            ,const CUDA_glutton_max_stack & p_stack
                            )
    {
        print_position_info(p_indent_level, p_stack, &CUDA_glutton_max_stack::get_position_info);
    }

    __device__
    void print_best_candidate_info(unsigned int p_indent_level
                                  ,const CUDA_glutton_max_stack & p_stack
                                  )
    {
        print_position_info(p_indent_level, p_stack, &CUDA_glutton_max_stack::get_best_candidate_info);
    }

    /**
     * Print information relating info index and position index
     * @param p_indent_level indentation level
     * @param p_stack
     */
    __device__
    void
    print_device_info_position_index(unsigned int p_indent_level
                                    ,const CUDA_glutton_max_stack & p_stack
                                    )
    {
        print_single(p_indent_level, "====== Position index <-> Info index ======\n");
        for(unsigned int l_index = 0; l_index <= (p_stack.get_nb_pieces() / 32); ++l_index)
        {
            unsigned int l_thread_index = 32 * l_index + threadIdx.x;
            print_mask(p_indent_level, __ballot_sync(0xFFFFFFFF, l_thread_index < p_stack.get_nb_pieces()), "Position[%" PRIu32 "] -> Index %" PRIu32 , l_thread_index, l_thread_index < p_stack.get_nb_pieces() ? p_stack.get_info_index(l_thread_index) : 0xDEADCAFE);
        }
        for(unsigned int l_index = 0; l_index <= (p_stack.get_size() / 32); ++l_index)
        {
            unsigned int l_thread_index = 32 * l_index + threadIdx.x;
            print_mask(p_indent_level, __ballot_sync(0xFFFFFFFF, l_thread_index < p_stack.get_size()), "%c Index[%" PRIu32 "] -> Position %" PRIu32 , l_thread_index < p_stack.get_size() - p_stack.get_level() ? '*' : ' ', l_thread_index, l_thread_index < p_stack.get_size() ? p_stack.get_position_index(l_thread_index) : 0xDEADCAFE);
        }
    }

    /**
     * Print information relating info index and position index
     * @param p_indent_level indentation level
     * @param p_stack
     */
    void
    print_host_info_position_index(unsigned int p_indent_level
                                  ,const CUDA_glutton_max_stack & p_stack
                                  )
    {
        std::cout << std::string(p_indent_level,' ') <<  "====== Position index <-> Info index ======" << std::endl;
        for(unsigned int l_index = 0; l_index < p_stack.get_nb_pieces(); ++l_index)
        {
            std::cout << std::string(p_indent_level,' ') << "Position[" << l_index << "] -> Index " << p_stack.get_info_index(l_index) << std::endl;
        }
        for(unsigned int l_index = 0; l_index < p_stack.get_size(); ++l_index)
        {
            std::cout << std::string(p_indent_level,' ') << (l_index < (p_stack.get_size() - p_stack.get_level()) ? '*' : ' ') << " Index[" << l_index << "] -> Position " << p_stack.get_position_index(l_index) << std::endl;
        }
    }

    __global__
    void kernel(CUDA_glutton_max_stack * p_stacks
               ,unsigned int p_nb_stack
               ,const CUDA_color_constraints & p_color_constraints
               )
    {
        assert(warpSize == blockDim.x);

        unsigned int l_stack_index = threadIdx.y + blockIdx.x * blockDim.y;

        if(l_stack_index >= p_nb_stack)
        {
            return;
        }

        CUDA_glutton_max_stack & l_stack = p_stacks[l_stack_index];

        bool l_new_level = true;
        unsigned int l_best_start_index = 0;

        while(l_stack.get_level() < l_stack.get_size())
        {

            print_single(0,"Stack level = %i", l_stack.get_level());


            if(l_new_level)
            {
                print_single(0,"Search for best score");
                uint32_t l_best_total_score = 0;
                uint32_t l_best_min_max_score = 0;
                unsigned int l_best_last_index = 0;

                // Iterate on all level position information to compute the score of each available transition
                for(unsigned int l_info_index = 0; l_info_index < l_stack.get_level_nb_info(); ++l_info_index)
                {
                    print_single(1,"Info index = %i <=> Position = %i", l_info_index, l_stack.get_position_index(l_info_index));

                    // At the beginning all threads participates to ballot
                    unsigned int l_ballot_result = 0xFFFFFFFF;

                    // Each thread get its word in position info
                    uint32_t l_thread_available_variables = l_stack.get_position_info(l_info_index).get_word(threadIdx.x);

                    print_all(2,"Thread available variables = 0x%" PRIx32, l_thread_available_variables);

                    // Iterate on non null position info words determined by ballot between threads
                    do
                    {
                        // Sync between threads to determine who as some available variables
                        l_ballot_result = __ballot_sync(l_ballot_result, (int) l_thread_available_variables);

                        print_mask(3, l_ballot_result, "Thread available variables = 0x%" PRIx32, l_thread_available_variables);

                        // Ballot result cannot be NULL because we are by construction in a valid situation
                        assert(l_ballot_result);

                        // Determine first lane/thread having an available variable. Result is greater than 0 due to assert
                        unsigned l_elected_thread = __ffs((int)l_ballot_result) - 1;

                        print_single(3, "Elected thread : %i", l_elected_thread);

                        // Eliminate thread from next ballot
                        l_ballot_result &= ~(1u << l_elected_thread);

                        // Copy available variables because we will iterate on it
                        uint32_t l_current_available_variables = l_thread_available_variables;

                        // Share current available variables with all other threads so they can select the same variable
                        l_current_available_variables = __shfl_sync(0xFFFFFFFF, l_current_available_variables, (int)l_elected_thread);

                        // Iterate on available variables of elected thread
                        do
                        {
                            print_single(4, "Current available variables : 0x%" PRIx32, l_current_available_variables);

                            // Determine first available variable. Result  cannot be 0 due to ballot
                            unsigned l_bit_index = __ffs((int)l_current_available_variables) - 1;

                            print_single(4, "Bit index : %i", l_bit_index);

                            // Set variable bit to zero
                            uint32_t l_mask = ~(1u << l_bit_index);
                            l_current_available_variables &= l_mask;

                            // Compute piece index
                            uint32_t l_piece_index = CUDA_piece_position_info2::compute_piece_index(l_elected_thread, l_bit_index);

                            print_single(4, "Piece index : %i", l_piece_index);

                            // Piece orientation
                            uint32_t l_piece_orientation = CUDA_piece_position_info2::compute_orientation_index(l_elected_thread, l_bit_index);

                            print_single(4, "Piece orientation : %i", l_piece_orientation);

                            // Get position index corresponding to this info index
                            uint32_t l_position_index = l_stack.get_position_index(l_info_index);

                            bool l_invalid = false;

                            uint32_t l_info_bits_min = 0xFFFFFFFFu;
                            uint32_t l_info_bits_max = 0;
                            uint32_t l_info_bits_total = 0;

                            if(!threadIdx.x)
                            {
                                l_stack.set_piece_unavailable(l_piece_index);
                            }
                            __syncwarp(0xFFFFFFFF);
                            l_stack.clear_piece_info();
                            CUDA_glutton_max_stack::t_piece_infos & l_piece_infos = l_stack.get_thread_piece_info();

                            uint32_t l_mask_to_apply = l_elected_thread == threadIdx.x ? (~CUDA_piece_position_info2::compute_piece_mask(l_bit_index)): 0xFFFFFFFFu;

                            // Each thread store the related info index corresponding to the orientation index
                            unsigned int l_related_thread_index = 0xFFFFFFFFu;

                            // Apply color constraint
                            print_single(4, "Apply color constraints");
                            for(unsigned int l_orientation_index = 0; l_orientation_index < 4; ++l_orientation_index)
                            {
                                uint32_t l_color_id = g_pieces[l_piece_index][(l_orientation_index + l_piece_orientation) % 4];
                                if(l_color_id)
                                {
                                    // Compute position index related to piece side
                                    uint32_t l_related_position_index = l_position_index + g_position_offset[l_orientation_index];

                                    // Check if position is free, if this not the case there is no corresponding index
                                    if(!l_stack.is_position_free(l_related_position_index))
                                    {
                                        print_single(5, "Position %i is not free:\n", l_related_position_index);
                                        continue;
                                    }

                                    // Compute corresponding info index
                                    uint32_t l_related_info_index = l_stack.get_info_index(l_related_position_index);
                                    print_single(5, "Info %i:\n", l_related_info_index);

                                    // Each thread store the related info index corresponding to the orientation index
                                    l_related_thread_index = threadIdx.x == l_orientation_index ? l_related_info_index : l_related_thread_index;

                                    uint32_t l_capability = l_stack.get_position_info(l_related_info_index).get_word(threadIdx.x);
                                    uint32_t l_constraint_capability = p_color_constraints.get_info(l_color_id - 1, l_orientation_index).get_word(threadIdx.x);
                                    l_constraint_capability &= l_mask_to_apply;

                                    //print_all(5, "Capability 0x%08" PRIx32 "\nConstraint 0x%08" PRIx32 "\n", l_capability, l_constraint_capability);
                                    if((l_invalid = analyze_info(l_capability, l_constraint_capability, l_info_bits_min, l_info_bits_max, l_info_bits_total, l_piece_infos)))
                                    {
                                        break;
                                    }
                                    //print_all(5, "Min %3i Max %3i Total %i\n", l_info_bits_min, l_info_bits_max, l_info_bits_total);
                                    print_mask(5, __ballot_sync(0xFFFFFFFFu, l_capability | l_constraint_capability), "Capability 0x%08" PRIx32 "\nConstraint 0x%08" PRIx32 "\nMin %3i\tMax %3i\tTotal %i\n", l_capability, l_constraint_capability, l_info_bits_min, l_info_bits_max, l_info_bits_total);
                                }
                            }
                            if(!l_invalid)
                            {
                                print_single(4, "Apply piece constraints before selected index");
                                for(unsigned int l_result_info_index = 0; l_result_info_index < l_info_index; ++l_result_info_index)
                                {
                                    if(__all_sync(0xFFFFFFFFu, l_result_info_index != l_related_thread_index))
                                    {
                                        print_single(5, "Info %i:\n", l_result_info_index);
                                        uint32_t l_capability = l_stack.get_position_info(l_result_info_index).get_word(threadIdx.x);
                                        if((l_invalid = analyze_info(l_capability, l_mask_to_apply, l_info_bits_min, l_info_bits_max, l_info_bits_total, l_piece_infos)))
                                        {
                                            break;
                                        }
                                        print_mask(5, __ballot_sync(0xFFFFFFFFu, l_capability), "Capability 0x%08" PRIx32 "\nConstraint 0x%08" PRIx32 "\nMin %3i\tMax %3i\tTotal %i\n", l_capability, l_mask_to_apply, l_info_bits_min, l_info_bits_max, l_info_bits_total);
                                    }
                                }
                            }
                            if(!l_invalid)
                            {
                                print_single(4, "Apply piece constraints after selected index");
                                for(unsigned int l_result_info_index = l_info_index + 1; l_result_info_index < l_stack.get_level_nb_info(); ++l_result_info_index)
                                {
                                    if(__all_sync(0xFFFFFFFFu, l_result_info_index != l_related_thread_index))
                                    {
                                        print_single(5, "Info %i:\n", l_result_info_index);
                                        uint32_t l_capability = l_stack.get_position_info(l_result_info_index).get_word(threadIdx.x);
                                        if((l_invalid = analyze_info(l_capability, l_mask_to_apply, l_info_bits_min, l_info_bits_max, l_info_bits_total, l_piece_infos)))
                                        {
                                            break;
                                        }
                                        print_mask(5, __ballot_sync(0xFFFFFFFFu, l_capability), "Capability 0x%08" PRIx32 "\nConstraint 0x%08" PRIx32 "\nMin %3i\tMax %3i\tTotal %i\n", l_capability, l_mask_to_apply, l_info_bits_min, l_info_bits_max, l_info_bits_total);
                                    }
                                }
                            }
                            // Manage pieces info
                            if(!l_invalid)
                            {
                                uint32_t l_piece_info_total_bit = 0;
                                uint32_t l_piece_info_min_bits = 0xFFFFFFFFu;
                                uint32_t l_piece_info_max_bits = 0;
                                for(unsigned int l_piece_info_index = 0; l_piece_info_index < 8; ++l_piece_info_index)
                                {
                                    CUDA_glutton_max_stack::t_piece_info l_piece_info = l_piece_infos[l_piece_info_index];
                                    if(__all_sync(0xFFFFFFFFu, l_piece_info))
                                    {
                                        unsigned int l_info_piece_index = 8 * threadIdx.x + l_piece_info_index;
                                        if(l_stack.is_piece_available(l_info_piece_index))
                                        {
                                            update_stats(l_piece_info, l_piece_info_min_bits, l_piece_info_max_bits, l_piece_info_total_bit);
                                            print_all(5, "Piece %i:\nMin %3i\tMax %3i\tTotal %i\n", l_info_piece_index, l_piece_info_min_bits, l_piece_info_max_bits, l_piece_info_total_bit);
                                        }
                                    }
                                    else
                                    {
                                        l_invalid = true;
                                        break;
                                    }
                                }
                                if(!l_invalid)
                                {
                                    l_info_bits_total += reduce_add_sync(l_piece_info_total_bit);
                                    l_piece_info_min_bits = reduce_min_sync(l_piece_info_min_bits);
                                    l_info_bits_min = l_piece_info_min_bits < l_info_bits_min ? l_piece_info_min_bits : l_info_bits_min;
                                    l_piece_info_max_bits = reduce_max_sync(l_piece_info_max_bits);
                                    l_info_bits_max = l_piece_info_max_bits > l_info_bits_max ? l_piece_info_max_bits : l_info_bits_max;
                                    print_single(4, "After reduction");
                                    print_single(4, "Min %3i\tMax %3i\tTotal %i\n", l_info_bits_min, l_info_bits_max, l_info_bits_total);
                                }
                            }
                            if(!l_invalid)
                            {
                                // compare with global stats
                                uint32_t l_min_max_score = (l_info_bits_max << 16u) + l_info_bits_min;
                                print_single(4, "Total %i\tMinMax %i\n", l_info_bits_total, l_min_max_score);
                                bool l_record_candidate = false;
                                if(l_info_bits_total > l_best_total_score || (l_info_bits_total == l_best_total_score && l_min_max_score > l_best_min_max_score))
                                {
                                    print_single(4, "New best score Total %i MinMax %i\n", l_info_bits_total, l_min_max_score);
                                    l_best_total_score = l_info_bits_total;
                                    l_best_min_max_score = l_min_max_score;
                                    // Clear previous candidate for best score
                                    for(unsigned int l_clear_info_index = l_best_start_index; l_clear_info_index <= l_best_last_index; ++l_clear_info_index)
                                    {
                                        // Clear previous candidate capability
                                        l_stack.get_best_candidate_info(l_clear_info_index).set_word(threadIdx.x, 0);
                                    }
                                    l_best_start_index = l_info_index;
                                    l_best_last_index = l_info_index;
                                    l_record_candidate = true;
                                }
                                else if(l_info_bits_total == l_best_total_score && l_min_max_score == l_best_min_max_score)
                                {
                                    print_single(4, "Same best score Total %i MinMax %i\n", l_info_bits_total, l_min_max_score);
                                    l_best_last_index = l_info_index;
                                    l_record_candidate = true;
                                }
                                if(l_record_candidate && !threadIdx.x)
                                {
                                    l_stack.get_best_candidate_info(l_info_index).set_bit(l_piece_index, static_cast<emp_types::t_orientation>(l_piece_orientation));
                                }
                                __syncwarp(0xFFFFFFFF);
                            }
                            if(!threadIdx.x)
                            {
                                l_stack.set_piece_available(l_piece_index);
                            }
                            __syncwarp(0xFFFFFFFF);
                        }  while(l_current_available_variables);

                    } while(l_ballot_result);
                }

                // If no best score found there is no interesting transition so go back
                if(!l_best_total_score)
                {
                    print_single(0, "No best score found, go up from one level");
                    print_device_info_position_index(0, l_stack);
                    l_best_start_index = l_stack.pop();
                    print_device_info_position_index(0, l_stack);
                    l_new_level = false;
                    continue;
                }
                // TO DELETE l_stack.unmark_best_candidates();
            }


            // At the beginning all threads participates to ballot
            unsigned int l_ballot_result = 0xFFFFFFFF;
            unsigned int l_best_candidate_index = l_best_start_index;
            uint32_t l_thread_best_candidates;

            print_single(0, "Iterate on best candidate from index %i", l_best_candidate_index);
            // Iterate on best candidates to prepare next level until we find a
            // candidate of reach the end of candidate info
            do
            {
                print_single(1,"Best Info index = %i <=> Position = %i", l_best_candidate_index, l_stack.get_position_index(l_best_candidate_index));

                // Each thread get its word in position info
                l_thread_best_candidates = l_stack.get_best_candidate_info(l_best_candidate_index).get_word(threadIdx.x);

                print_all(1,"Thread best candidates = 0x%" PRIx32, l_thread_best_candidates);

                // Sync between threads to determine who as some available variables
                l_ballot_result = __ballot_sync(l_ballot_result, (int) l_thread_best_candidates);

                print_mask(1, l_ballot_result, "Thread best candidates = 0x%" PRIx32, l_thread_best_candidates);

                // Ballot result cannot be NULL because we are by construction in a valid situation
                if(l_ballot_result)
                {
                    break;
                }
                ++l_best_candidate_index;

            } while(l_best_candidate_index < l_stack.get_level_nb_info());

            // No candidate found so we go up from one level
            if(l_best_candidate_index == l_stack.get_level_nb_info())
            {
                print_single(0, "No more best score, go up from one level");
                print_device_info_position_index(0, l_stack);
                l_best_start_index = l_stack.pop();
                print_device_info_position_index(0, l_stack);
                l_new_level = false;
                continue;
            }

            assert(l_ballot_result);

            // Determine first lane/thread having a candidate. Result is greater than 0 due to assert
            unsigned l_elected_thread = __ffs((int)l_ballot_result) - 1;

            print_single(0, "Elected thread : %i", l_elected_thread);

            // Share current best candidate with all other threads so they can select the same candidate
            l_thread_best_candidates = __shfl_sync(0xFFFFFFFF, l_thread_best_candidates, (int)l_elected_thread);

            // Determine first available candidate. Result  cannot be 0 due to ballot result
            unsigned l_bit_index = __ffs((int)l_thread_best_candidates) - 1;

            print_single(0, "Bit index : %i", l_bit_index);

            print_position_info(6, l_stack);

            // Set variable bit to zero in best candidate and current info
            if(threadIdx.x < 2)
            {
                CUDA_piece_position_info2 & l_position_info = threadIdx.x ? l_stack.get_best_candidate_info(l_best_candidate_index) : l_stack.get_position_info(l_best_candidate_index);
                l_position_info.clear_bit(l_elected_thread, l_bit_index);
            }
            __syncwarp(0xFFFFFFFF);
            print_single(0, "after clear\n");
            print_position_info(6, l_stack);

            // Compute piece index
            uint32_t l_piece_index = CUDA_piece_position_info2::compute_piece_index(l_elected_thread, l_bit_index);

            print_single(0, "Piece index : %i", l_piece_index);

            // Piece orientation
            uint32_t l_piece_orientation = CUDA_piece_position_info2::compute_orientation_index(l_elected_thread, l_bit_index);

            print_single(0, "Piece orientation : %i", l_piece_orientation);

            // Get position index corresponding to this info index
            uint32_t l_position_index = l_stack.get_position_index(l_best_candidate_index);

            {
                // Compute mask to apply which set piece bit to 0
                uint32_t l_mask_to_apply = l_elected_thread == threadIdx.x ? (~CUDA_piece_position_info2::compute_piece_mask(l_bit_index)): 0xFFFFFFFFu;
                for (unsigned int l_result_info_index = 0; l_result_info_index < l_best_candidate_index; ++l_result_info_index)
                {
                    print_single(1, "Info %i -> %i:\n", l_result_info_index, l_result_info_index);
                    uint32_t l_capability = l_stack.get_position_info(l_result_info_index).get_word(threadIdx.x);
                    uint32_t l_constraint = l_mask_to_apply;
                    uint32_t l_result = l_capability & l_constraint;
                    print_mask(1, __ballot_sync(0xFFFFFFFFu, l_capability), "Capability 0x%08" PRIx32 "\nConstraint 0x%08" PRIx32 "\nResult     0x%08" PRIx32 "\n", l_capability, l_mask_to_apply, l_result);
                    l_stack.get_next_level_position_info(l_result_info_index).set_word(threadIdx.x, l_result);
                }

                // Last position is not treated here because next level has 1 position less
                for (unsigned int l_result_info_index = l_best_candidate_index + 1; l_result_info_index < l_stack.get_level_nb_info() - 1; ++l_result_info_index)
                {
                    print_single(1, "Info %i -> %i:\n", l_result_info_index, l_result_info_index);
                    uint32_t l_capability = l_stack.get_position_info(l_result_info_index).get_word(threadIdx.x);
                    uint32_t l_constraint = l_mask_to_apply;
                    uint32_t l_result = l_capability & l_constraint;
                    print_mask(1, __ballot_sync(0xFFFFFFFFu, l_capability), "Capability 0x%08" PRIx32 "\nConstraint 0x%08" PRIx32 "\nResult     0x%08" PRIx32 "\n", l_capability, l_mask_to_apply, l_result);
                    l_stack.get_next_level_position_info(l_result_info_index).set_word(threadIdx.x, l_result);
                }

                // Last position in next level it will be located at l_best_candidate_index
                print_single(0, "Info %i -> %i:\n", l_stack.get_level_nb_info() - 1, l_best_candidate_index);
                uint32_t l_capability = l_stack.get_position_info(l_stack.get_level_nb_info() - 1).get_word(threadIdx.x);
                uint32_t l_constraint = l_mask_to_apply;
                uint32_t l_result = l_capability & l_constraint;
                print_mask(1, __ballot_sync(0xFFFFFFFFu, l_capability), "Capability 0x%08" PRIx32 "\nConstraint 0x%08" PRIx32 "\nResult     0x%08" PRIx32 "\n", l_capability, l_mask_to_apply, l_result);
                l_stack.get_next_level_position_info(l_best_candidate_index).set_word(threadIdx.x , l_result);

                print_device_info_position_index(0, l_stack);
                l_stack.push(l_best_candidate_index, l_position_index, l_piece_index, l_piece_orientation);
                print_device_info_position_index(0, l_stack);

                // Print relation index to position
                for(unsigned int l_warp_index = 0; l_warp_index <= l_stack.get_size() / 32; ++l_warp_index)
                {
                    unsigned int l_index = 32 * l_warp_index + threadIdx.x;
                    bool l_valid_index = l_index < l_stack.get_size() && l_stack.is_position_index_used(l_index);
                    print_mask(1, __ballot_sync(0xFFFFFFFFu, l_valid_index), "Index %i -> Position %i", l_index, l_valid_index ? l_stack.get_position_index(l_index) : 0xDEADCAFEu);
                }

                // Apply color constraint
                for(unsigned int l_orientation_index = 0; l_orientation_index < 4; ++l_orientation_index)
                {
                    uint32_t l_color_id = g_pieces[l_piece_index][(l_orientation_index + l_piece_orientation) % 4];
                    print_single(1, "Color Id %i", l_color_id);
                    if(l_color_id)
                    {
                        // Compute position index related to piece side
                        uint32_t l_related_position_index = l_position_index + g_position_offset[l_orientation_index];
                        print_single(1, "Related position index %i", l_related_position_index);

                        // Check if position is free, if this not the case there is no corresponding index
                        if(!l_stack.is_position_free(l_related_position_index))
                        {
                            print_single(1,"Position %i is not free", l_related_position_index);
                            continue;
                        }

                        // Compute corresponding info index
                        uint32_t l_related_info_index = l_stack.get_info_index(l_related_position_index);
                        print_single(1, "Related info index %i", l_related_info_index);

                        // If related index correspond to last position than result is stored in postition where we store the piece
                        uint32_t l_related_target_info_index = l_related_info_index < l_stack.get_level_nb_info() - 1 ? l_related_info_index : l_position_index;

                        print_single(1, "Color Info %i -> %i:\n", l_related_info_index, l_related_target_info_index);
                        print_mask(1, __ballot_sync(0xFFFFFFFFu, l_stack.get_position_info(l_related_info_index).get_word(threadIdx.x) | p_color_constraints.get_info(l_color_id - 1, l_orientation_index).get_word(threadIdx.x)), "Capability 0x%08" PRIx32 "\nConstraint 0x%08" PRIx32 "\nResult 0x%08" PRIx32 "\n", l_stack.get_position_info(l_related_info_index).get_word(threadIdx.x), p_color_constraints.get_info(l_color_id - 1, l_orientation_index).get_word(threadIdx.x),l_stack.get_position_info(l_related_info_index).get_word(threadIdx.x) & p_color_constraints.get_info(l_color_id - 1, l_orientation_index).get_word(threadIdx.x));
                        l_stack.get_position_info(l_related_target_info_index).CUDA_and(l_stack.get_position_info(l_related_info_index), p_color_constraints.get_info(l_color_id - 1, l_orientation_index));
                    }
                }
            }

            // For latest level we do not search for best score at is zero in any case
            l_new_level = l_stack.get_level() < (l_stack.get_size() - 1);
        }

        print_single(0, "End with stack level %i", l_stack.get_level());
    }

    //-------------------------------------------------------------------------
    void launch_CUDA_glutton_max(const emp_piece_db & p_piece_db
                                ,const emp_FSM_info & p_info
                                )
    {
        // Prepare piece description
        std::array<uint32_t, 256 * 4> l_pieces{};
        for(unsigned int l_piece_index = 0; l_piece_index < p_info.get_nb_pieces(); ++l_piece_index)
        {
            for(auto l_orientation: emp_types::get_orientations())
            {
                l_pieces[l_piece_index * 4 + static_cast<unsigned int>(l_orientation)] = p_piece_db.get_piece(l_piece_index + 1).get_color(l_orientation);
            }
        }

        // Prepare position offset
        std::array<int,4> l_x_offset{- static_cast<int>(p_info.get_width()), 1, static_cast<int>(p_info.get_width()), -1};
        unsigned int l_nb_pieces = p_info.get_nb_pieces();

        CUDA_info();

        // Fill constant variables
        hipMemcpyToSymbol(HIP_SYMBOL(g_pieces), l_pieces.data(), l_pieces.size() * sizeof(uint32_t ));
        hipMemcpyToSymbol(HIP_SYMBOL(g_position_offset), l_x_offset.data(), l_x_offset.size() * sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(g_nb_pieces), &l_nb_pieces, sizeof(unsigned int));

        // Prepare color constraints
        CUDA_piece_position_info2::set_init_value(0);
        std::unique_ptr<CUDA_color_constraints> l_color_constraints{new CUDA_color_constraints(static_cast<unsigned int>(p_piece_db.get_colors().size()))};
        for(auto l_iter_color: p_piece_db.get_colors())
        {
            unsigned int l_color_index = l_iter_color - 1;
            for(auto l_color_orientation: emp_types::get_orientations())
            {
                auto l_opposite_orientation = emp_types::get_opposite(l_color_orientation);
                for(unsigned int l_piece_index = 0; l_piece_index < p_info.get_nb_pieces(); ++l_piece_index)
                {
                    for(auto l_piece_orientation: emp_types::get_orientations())
                    {
                        emp_types::t_color_id l_color_id{p_piece_db.get_piece(l_piece_index + 1).get_color(l_opposite_orientation, l_piece_orientation)};
                        if(l_color_id == l_iter_color)
                        {
                            l_color_constraints->get_info(l_color_index, static_cast<unsigned int>(l_color_orientation)).set_bit(l_piece_index, l_piece_orientation);
                        }
                    }
                }
                std::cout << "Color " << l_iter_color << emp_types::orientation2short_string(l_color_orientation) << ":" << std::endl;
                std::cout << l_color_constraints->get_info(l_color_index, static_cast<unsigned int>(l_color_orientation)) << std::endl;
            }
        }

        // Prepare initial situation vector
        CUDA_piece_position_info2::set_init_value(0x0);
        auto * l_initial_capability = new CUDA_piece_position_info2[p_info.get_nb_pieces()];
        for(unsigned int l_position_index = 0; l_position_index < p_info.get_nb_pieces(); ++l_position_index)
        {
            switch(p_info.get_position_kind(p_info.get_x(l_position_index), p_info.get_y(l_position_index)))
            {
                case emp_types::t_kind::CORNER:
                {
                    emp_types::t_orientation l_border1;
                    emp_types::t_orientation l_border2;
                    std::tie(l_border1,l_border2) = p_info.get_corner_orientation(l_position_index);
                    for (unsigned int l_corner_index = 0; l_corner_index < 4; ++l_corner_index)
                    {
                        const emp_piece_corner & l_corner = p_piece_db.get_corner(l_corner_index);
                        l_initial_capability[l_position_index].set_bit(l_corner.get_id() - 1, l_corner.compute_orientation(l_border1, l_border2));
                    }
                }
                    break;
                case emp_types::t_kind::BORDER:
                {
                    emp_types::t_orientation l_border_orientation = p_info.get_border_orientation(l_position_index);
                    for(unsigned int l_border_index = 0; l_border_index < p_info.get_nb_borders(); ++l_border_index)
                    {
                        const emp_piece_border & l_border = p_piece_db.get_border(l_border_index);
                        l_initial_capability[l_position_index].set_bit(l_border.get_id() - 1, l_border.compute_orientation(l_border_orientation));
                    }
                }
                    break;
                case emp_types::t_kind::CENTER:
                    for(unsigned int l_center_index = 0; l_center_index < p_info.get_nb_centers(); ++l_center_index)
                    {
                        const emp_piece & l_center = p_piece_db.get_center(l_center_index);
                        for (auto l_iter: emp_types::get_orientations())
                        {
                            l_initial_capability[l_position_index].set_bit(l_center.get_id() - 1, l_iter);
                        }
                    }
                    break;
                case emp_types::t_kind::UNDEFINED:
                    throw quicky_exception::quicky_logic_exception("Undefined position type", __LINE__, __FILE__);
                default:
                    throw quicky_exception::quicky_logic_exception("Unknown position type", __LINE__, __FILE__);
            }
        }

        for(unsigned int l_position_index = 0; l_position_index < p_info.get_nb_pieces(); ++l_position_index)
        {
            std::cout << "Position " << l_position_index << "(" << p_info.get_x(l_position_index) << "," <<p_info.get_y(l_position_index) << "):" << std::endl;
            std::cout << l_initial_capability[l_position_index] << std::endl;
        }
        std::unique_ptr<CUDA_memory_managed_array<uint32_t>> l_cuda_array{new CUDA_memory_managed_array<uint32_t>(32)};
        for(unsigned int l_index = 0; l_index < 32 ; ++l_index)
        {
            (*l_cuda_array)[l_index] = 0;
        }

        emp_situation l_start_situation;

        unsigned int l_size = l_nb_pieces - l_start_situation.get_level();
        std::unique_ptr<CUDA_glutton_max_stack> l_stack{new CUDA_glutton_max_stack(l_size,l_nb_pieces)};
        for(unsigned int l_piece_index = 0; l_piece_index < l_nb_pieces; ++l_piece_index)
        {
            l_stack->set_piece_available(l_piece_index);
        }

        // Prepare stack with info of initial situation
        uint32_t l_info_index = 0;
        for(unsigned int l_position_index = 0; l_position_index < l_nb_pieces; ++l_position_index)
        {
            unsigned int l_x = p_info.get_x(l_position_index);
            unsigned int l_y = p_info.get_y(l_position_index);
            if(!l_start_situation.contains_piece(l_x, l_y))
            {
                l_stack->set_position_info_relation(l_info_index, l_position_index);
                l_stack->set_position_info(l_info_index, l_initial_capability[l_position_index]);
                ++l_info_index;
            }
            else
            {
                l_stack->set_piece_unavailable(l_start_situation.get_piece(l_x, l_y).first - 1);
            }
        }
        delete[] l_initial_capability;
        print_host_info_position_index(0, *l_stack);

        // Reset CUDA error status
        hipGetLastError();
        std::cout << "Launch kernels" << std::endl;
        dim3 l_block_info(32, 1);
        dim3 l_grid_info(1);
        kernel<<<l_grid_info, l_block_info>>>(l_stack.get(), 1, *l_color_constraints);
        //test_kernel<<<l_grid_info, l_block_info>>>(l_stack.get(), 1, *l_cuda_array);
        hipDeviceSynchronize();
        gpuErrChk(hipGetLastError());

        if(l_stack->is_empty())
        {
            std::cout << "Empty stack" << std::endl;
        }
        else
        {
            unsigned int l_max_level = l_stack->get_level() - (unsigned int)l_stack->is_full();
            for(unsigned int l_level = 0; l_level <= l_max_level; ++l_level)
            {
                CUDA_glutton_max_stack::played_info_t l_played_info = l_stack->get_played_info(l_level);
                unsigned int l_x = p_info.get_x(CUDA_glutton_max_stack::decode_position_index(l_played_info));
                unsigned int l_y = p_info.get_y(CUDA_glutton_max_stack::decode_position_index(l_played_info));
                assert(!l_start_situation.contains_piece(l_x, l_y));
                l_start_situation.set_piece(l_x
                                           ,l_y
                                           ,emp_types::t_oriented_piece{static_cast<emp_types::t_piece_id >(1 + CUDA_glutton_max_stack::decode_piece_index(l_played_info))
                                                                       ,static_cast<emp_types::t_orientation>(CUDA_glutton_max_stack::decode_orientation_index(l_played_info))
                                                                       }
                                           );
            }
            std::cout << "Situation with stack played info:" << std::endl;
            std::cout << situation_string_formatter<emp_situation>::to_string(l_start_situation) << std::endl;
        }
        for(unsigned int l_index = 0; l_index < l_size; ++l_index)
        {
            std::cout << l_stack->get_position_info(l_index) << std::endl;
            //l_stack->push();
        }

        std::cout << "CUDA array content" << std::endl;
        for(unsigned int l_index = 0; l_index < 32; ++l_index)
        {
            std::cout << "cuda_array[" << l_index << "] = " << (*l_cuda_array)[l_index] << std::endl;
        }

    }

    //-------------------------------------------------------------------------
    __global__
    void test_kernel(CUDA_glutton_max_stack * p_stacks
                    ,unsigned int p_nb_stack
                    ,CUDA_memory_managed_array<uint32_t> & p_array
                    )
    {
        assert(warpSize == blockDim.x);

        p_array[threadIdx.x] = 31 -  threadIdx.x;

        print_all(0, "managed_array[%i] = %i", threadIdx.x, p_array[threadIdx.x]);

        for(unsigned int l_piece_index = 0; l_piece_index < g_nb_pieces; ++l_piece_index)
        {
            print_single(1, "Piece[%i]={%i, %i, %i, %i}\n", l_piece_index + 1, g_pieces[l_piece_index][0], g_pieces[l_piece_index][1], g_pieces[l_piece_index][2], g_pieces[l_piece_index][3]);
        }
        return;
        unsigned int l_stack_index = threadIdx.y + blockIdx.x * blockDim.y;

        if(l_stack_index >= p_nb_stack)
        {
            return;
        }

        CUDA_glutton_max_stack & l_stack = p_stacks[l_stack_index];


        for(unsigned int l_index = 0; l_index < l_stack.get_size(); ++l_index)
        {
            l_stack.get_position_info(l_index);
            l_stack.push(0,0,0,0);
        }
        for(unsigned int l_index = 0; l_index < l_stack.get_size(); ++l_index)
        {
            l_stack.pop();
        }
    }

}
// EOF
